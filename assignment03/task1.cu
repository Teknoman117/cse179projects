#include "hip/hip_runtime.h"
#include "utilities.hpp"

// Task 1 kernel - simple, only uses global memory.
__global__ void GameOfLifeGPU(unsigned char *grid, unsigned char *resultGrid, size_t M, size_t N)
{
    // Get the global address of this thread
    ssize_t x = (blockIdx.x * blockDim.x) + threadIdx.x;
    ssize_t y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Compute the number of live neighbors
    short liveNeighbors = 0;

    // We manually check all of them for performance reasons
    ssize_t aX, aY;
    aX = x - 1; aY = y - 1;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    aX = x; aY = y - 1;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    aX = x + 1; aY = y - 1;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    aX = x - 1; aY = y;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    unsigned char localCellValue = grid[(y*M)+x];  // our value

    aX = x + 1; aY = y;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    aX = x - 1; aY = y + 1;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    aX = x; aY = y + 1;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    aX = x + 1; aY = y + 1;
    if(aX >= 0 && aY >= 0 && aX < M && aY < N)
        liveNeighbors += (grid[(aY * M) + aX]) ? 1 : 0;

    // Perform game of life logic
    if(localCellValue == CELL_STATUS_ALIVE)
    {
        // if we have one or two neighbors, we die from loneliness
        if(liveNeighbors < 2 || liveNeighbors > 3)
            localCellValue = CELL_STATUS_DEAD;
    }
    else
    {
        // If we have two or three neighbors, we LIVE
        if(liveNeighbors == 3)
            localCellValue = CELL_STATUS_ALIVE;
    }
    resultGrid[(y*M)+x] = localCellValue;
}

// Run the game of life experiment on the GPU
float GameOfLifeGPU_Experiment(unsigned char *grid, unsigned char *resultGrid, size_t M, size_t N, size_t B, size_t T, size_t iterations)
{
    // Allocate device memory for the experiment
    unsigned char *deviceGrid, *deviceGridResult;
    hipMalloc(&deviceGrid, M*N);
    hipMalloc(&deviceGridResult, M*N);
    hipMemcpy(deviceGrid, grid, M*N, hipMemcpyHostToDevice);

    // Compute block and grid sizes
    dim3 gridSize(B,B,1);
    dim3 blockSize(T,T,1);

    // Begin to record time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Perform all the iterations
    for(int i = 0; i < iterations; i++)
    {
        // Perform the iteration
        GameOfLifeGPU<<<gridSize,blockSize>>>(deviceGrid, deviceGridResult, M, N);

        // Swap the pointers
        unsigned char *temp = deviceGrid;
        deviceGrid = deviceGridResult;
        deviceGridResult = temp;
    }

    // GPU Execution time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);

    // Release memory
    hipMemcpy(resultGrid, deviceGrid, M*N, hipMemcpyDeviceToHost);
    hipFree(deviceGrid);
    hipFree(deviceGridResult);

    // Return
    return ms;
}
