#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

// Unrolled reduction operation for one warp
template <unsigned int blockSize>
__device__ void warpReduce(volatile float *mins, volatile float *maxs, volatile float *mean, volatile float *meansquares, unsigned int tid)
{
    if(blockSize >= 64) 
    {
        mins[tid] = (mins[tid] > mins[tid+32]) ? mins[tid+32] : mins[tid];
        maxs[tid] = (maxs[tid] < maxs[tid+32]) ? maxs[tid+32] : maxs[tid];
        mean[tid] += mean[tid+32];
        meansquares[tid] += meansquares[tid+32]*meansquares[tid+32];
    }
    if(blockSize >= 32) 
    {
        mins[tid] = (mins[tid] > mins[tid+16]) ? mins[tid+16] : mins[tid];
        maxs[tid] = (maxs[tid] < maxs[tid+16]) ? maxs[tid+16] : maxs[tid];
        mean[tid] += mean[tid+16];
        meansquares[tid] += meansquares[tid+16]*meansquares[tid+16];
    }
    if(blockSize >= 16) 
    {
        mins[tid] = (mins[tid] > mins[tid+8]) ? mins[tid+8] : mins[tid];
        maxs[tid] = (maxs[tid] < maxs[tid+8]) ? maxs[tid+8] : maxs[tid];
        mean[tid] += mean[tid+8];
        meansquares[tid] += meansquares[tid+8]*meansquares[tid+8];
    }
    if(blockSize >= 8) 
    {
        mins[tid] = (mins[tid] > mins[tid+4]) ? mins[tid+4] : mins[tid];
        maxs[tid] = (maxs[tid] < maxs[tid+4]) ? maxs[tid+4] : maxs[tid];
        mean[tid] += mean[tid+4];
        meansquares[tid] += meansquares[tid+4]*meansquares[tid+4];
    }
    if(blockSize >= 4) 
    {
        mins[tid] = (mins[tid] > mins[tid+2]) ? mins[tid+2] : mins[tid];
        maxs[tid] = (maxs[tid] < maxs[tid+2]) ? maxs[tid+2] : maxs[tid];
        mean[tid] += mean[tid+2];
        meansquares[tid] += meansquares[tid+2]*meansquares[tid+2];
    }
    if(blockSize >= 2) 
    {
        mins[tid] = (mins[tid] > mins[tid+1]) ? mins[tid+1] : mins[tid];
        maxs[tid] = (maxs[tid] < maxs[tid+1]) ? maxs[tid+1] : maxs[tid];
        mean[tid] += mean[tid+1];
        meansquares[tid] += meansquares[tid+1]*meansquares[tid+1];
    }
}

template <unsigned int blockSize>
__global__ void reduce(volatile float *inputMins, volatile float *inputMaxs, volatile float *inputMean, volatile float *meansquares, T *outputData, long N)
{
    // Shared data buffer for this block
    __shared__ float sharedData[sizeof(float)*blockSize];

    // Compute the
    unsigned int tid = threadIdx.x;
    unsigned long i = blockIdx.x*(blockSize*2) + tid;
    unsigned long gridSize = blockSize*2*gridDim.x;
    sharedData[tid] = 0;

    // All threads perform gather from global memory to into shared memory with an operation
    while(i < N)
    {
        float t = Operation(inputData[i], inputData[i+blockSize]);
        sharedData[tid] = Operation(sharedData[tid], t);
        i += gridSize;
    }
    __syncthreads();

    // Unrolled reduction (for greater than one warp active)
    if(blockSize >= 1024)
    {
        if(tid < 512) sharedData[tid] = Operation(sharedData[tid],sharedData[tid+512]);
        __syncthreads();
    }
    if(blockSize >= 512)
    {
        if(tid < 256) sharedData[tid] = Operation(sharedData[tid],sharedData[tid+256]);
        __syncthreads();
    }
    if(blockSize >= 256)
    {
        if(tid < 128) sharedData[tid] = Operation(sharedData[tid],sharedData[tid+128]);
        __syncthreads();
    }
    if(blockSize >= 128)
    {
        if(tid < 64) sharedData[tid] = Operation(sharedData[tid],sharedData[tid+64]);
        __syncthreads();
    }

    // Call unrolled reduction operation for a singular warp
    if(tid < 32) warpReduce<T,Operation,blockSize>(sharedData, tid);

    // Store result
    if(tid==0) outputData[blockIdx.x] = sharedData[0];
};

// Launch a reduction
template<typename T, T (*Operation)(const T a, const T b)>
void launchReduce (T *inputData, T *outputData, long N)
{
    // How many threads to we need
    long threadCount = (N/2 >= 1024) ? 1024 : N/2;

    // Compute the amount of elements per thread
    long elementCount = N / threadCount;

    // Limit the per thread element consumption to 1024
    long blockCount = 1;
    if(elementCount > 1024)
    {
        blockCount = elementCount / 1024;
        elementCount = 1024;
    }

    dim3 blockDim(blockCount);
    dim3 threadDim(threadCount);

    cout << "Lauching reduce<<<" << blockCount << "," << threadCount << ">>>(.,.," << N;
    cout << "), consuming " << elementCount << " elements per thread" << endl;

    // Launch
    switch (threadCount)
    {
        case 1024:
            reduce<T,Operation,1024><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 512:
            reduce<T,Operation,512><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 256:
            reduce<T,Operation,256><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 128:
            reduce<T,Operation,128><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 64:
            reduce<T,Operation,64><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 32:
            reduce<T,Operation,32><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 16:
            reduce<T,Operation,16><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 8:
            reduce<T,Operation,8><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 4:
            reduce<T,Operation,4><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 2:
            reduce<T,Operation,2><<<blockDim,threadDim>>>(inputData,outputData,N); break;
        case 1:
            reduce<T,Operation,1><<<blockDim,threadDim>>>(inputData,outputData,N); break;
    }
}


template <typename T>
__device__ T sumOperation (T a, T b)
{
    return a + b;
};

template<typename T>
__device__ T gtOperation (T a, T b)
{
    return (a > b) ? a : b;
};

template<typename T>
__device__ T ltOperation (T a, T b)
{
    return (a < b) ? a : b;
};

// Finds the maximum element in a set of numbers (N must be power of two)
template<typename T>
T getMaximumElementCUDA(T* data, long N)
{
    // Copy the data to the GPU
    float delta = 0.0f;
    T *deviceData = NULL;
    T *deviceResults = NULL;
    long resultsN = N;
    hipMalloc(&deviceData, N*sizeof(T));
    hipMemcpy(deviceData, data, N*sizeof(T), hipMemcpyHostToDevice);

    // Allocate a secondary buffer for storing results
    resultsN = (N / (1024*1024) > 1) ? N / (1024*1024) : 1;
    hipMalloc(&deviceResults, resultsN*sizeof(T));

    // Perform reduction
    do
    {
        // Compute the number of results
        resultsN = (N / (1024*1024) > 1) ? N / (1024*1024) : 1;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Run the reduction
        launchReduce<T,gtOperation>(deviceData,deviceResults,N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);
        delta += ms;
        printf(" >> Elapsed Time (GPU): %f ms\n", ms);

        // If resultsN is greater than one, swap arrays
        if(resultsN > 1)
        {
            T* temp = deviceResults;
            deviceResults = deviceData;
            deviceData = temp;
            N = resultsN;
        }

    } while (resultsN > 1);

    // Get the results
    int result = 0;
    hipMemcpy(&result, deviceResults, sizeof(T), hipMemcpyDeviceToHost);


    printf("Overall Elapsed Time (GPU): %f ms\n", delta);

    // Cleanup
    hipFree(deviceData);
    hipFree(deviceResults);
    return result;
}

template<typename T>
T getMaximumElementCPU(T* data, long N)
{
    T max = data[0];
    for(long i = 1; i < N; i++)
        if(data[i] > max) max = data[i];
    return max;
}


int main (int argc, char** argv)
{
    srand(time(NULL));

    int *local = NULL;
    long N = 536870912;
    hipHostMalloc(&local, N*sizeof(int));

    cout << "Generating Random Dataset" << endl;
    for(int i = 0; i < N; i++)
    {
        local[i] = rand();
    }
    cout << "Done" << endl << endl;


    cout << "Getting maximum on CPU" << endl;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int m = getMaximumElementCPU(local, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    cout << "Done: " << m  << " (elapsed: " << ms << " ms)" << endl << endl;

    cout << "Getting maximum on GPU" << endl;
    cout << "Done: " << getMaximumElementCUDA(local, N) << endl;

    hipHostFree(local);

    return 0;
}
